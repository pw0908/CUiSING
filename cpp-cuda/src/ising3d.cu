#include "hip/hip_runtime.h"
#include "ising.h"

/**************************/
/*      CUDA Kernels      */
/**************************/

/* __global__ void cudaMCIteration3dKernel
 *
 * Location: DEVICE
 *
 * This CUDA kernel is run on the device in order to run monte carlo 
 * updates in parallel. The kernel implements a checkerboard algorithm
 * to only attempts flips of non-interacting spins that can be flipped
 * in parallel. The attempted flips are accepted/rejected based on the
 * Metropolis criteria which uses the Boltzmann weight. The lattice is
 * updated directly on the device memory.
 * 
 * Periodic boundary conditions are considered. The thread id is
 * converted from linear indexing to 3d indexing, and then the mod
 * operator (%) is employed to wrap the lattice in both dimensions,
 * enforcing periodic boundary conditions.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - const float *__restrict__ rands: numbers generated from U(0,1), stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 * 
 * Output:
 *    - none (void)
 * 
 * */
template<int sub_lattice>
__global__ void cudaMcIteration3dKernel(signed int *lattice,
                                        const float *__restrict__ rands,
                                        const int n,
                                        const float J,
                                        const float h) {

    const long long tid = (long long)(blockDim.x)*blockIdx.x + threadIdx.x;
    const int i = tid/(n*n);
    const int j = (tid/n)%n;
    const int k = tid%n;
    
    if (tid >= n*n*n)
    {
        return;
    }
    else if ( ((k%2 == j%2) != i%2) != sub_lattice)
    {
        return;
    }
    
    const int sl = lattice[i*n*n+(n+(j-1)%n)%n*n+k];
    const int sr = lattice[i*n*n+(n+(j+1)%n)%n*n+k];
    const int su = lattice[(n+(i+1)%n)%n*n*n+j*n+k];
    const int sd = lattice[(n+(i-1)%n)%n*n*n+j*n+k];
    const int sn = lattice[i*n*n+j*n+(n+(k-1)%n)%n];
    const int ss = lattice[i*n*n+j*n+(n+(k+1)%n)%n];

    const int sum_spins = sl + sr + su + sd + sn + ss;
    const int sijk = lattice[tid];
    float boltz = exp(-2.0*sijk*(sum_spins*J+h));
    if (rands[tid]<=boltz) lattice[tid] = -sijk;
}


/* __global__ void cudaCalcHamiltonian2dKernel
 *
 * Location: DEVICE
 * 
 * This CUDA kernel is run on the device in order to calculate the
 * total Hamiltonian (energy) rapidly. The kernel utilizes shared
 * memory and a binary recution, followed by an atomic addition
 * to maximize the speed of the sum. The local energy of each spin
 * is calculated per thread.
 * 
 * Periodic boundary conditions are considered. The thread id is
 * converted from linear indexing to 2d indexing, and then the mod
 * operator (%) is employed to wrap the lattice in both dimensions,
 * enforcing periodic boundary conditions.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *E: energy trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
__global__ void cudaCalcHamiltonian3dKernel(signed int *lattice,
                                            float *E,
                                            const int n,
                                            const float J,
                                            const float h,
                                            const int iter) {
    
    extern __shared__ float shmem[];

    unsigned tid = threadIdx.x;
    unsigned idx = tid + blockIdx.x * blockDim.x;
    shmem[tid] = 0.0;

    unsigned i,j,k;
    int sl,sr,su,sd,sn,ss,sijk;
    for (; idx < n*n*n; idx += blockDim.x * gridDim.x)
    {
        i = idx/(n*n);
        j = (idx/n)%n;
        k = idx%n;

        sl = lattice[i*n*n+(n+(j-1)%n)%n*n+k];
        sr = lattice[i*n*n+(n+(j+1)%n)%n*n+k];
        su = lattice[(n+(i+1)%n)%n*n*n+j*n+k];
        sd = lattice[(n+(i-1)%n)%n*n*n+j*n+k];
        sn = lattice[i*n*n+j*n+(n+(k-1)%n)%n];
        ss = lattice[i*n*n+j*n+(n+(k+1)%n)%n];
        sijk = lattice[idx];
        
        shmem[tid] -= sijk*(J*(sl+sr+su+sd+sn+ss)/2.0+h);
    }
    __syncthreads();

    for (unsigned s = blockDim.x / 2; s>0; s>>=1)
    {
        if (tid < s)
            shmem[tid] += shmem[tid + s];
        
        __syncthreads();
    }

    if (threadIdx.x == 0)
        atomicAdd(&E[iter], shmem[0]/(3*n*n*n));
}


/* __global__ void cudaCalcMagnetization3dKernel
 *
 * Location: DEVICE
 * 
 * This CUDA kernel is run on the device in order to calculate the
 * total relative magnetization rapidly. The kernel utilizes shared
 * memory and a binary recution, followed by an atomic addition
 * to maximize the speed of the sum. Relative magnetization is the
 * sum of all spin values divided by the total number of spins.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *M: magnetization trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
__global__ void cudaCalcMagnetization3dKernel(signed int *lattice,
                                              float *M,
                                              const int n,
                                              const float J,
                                              const float h,
                                              const int iter) {
    
    extern __shared__ float shmem[];

    unsigned tid = threadIdx.x;
    unsigned idx = tid + blockIdx.x * blockDim.x;
    shmem[tid] = 0.0;
    for (; idx < n*n*n; idx += blockDim.x * gridDim.x)
    {
        shmem[tid] += lattice[idx];
    }
    __syncthreads();

    for (unsigned s = blockDim.x / 2; s>0; s>>=1)
    {
        if (tid < s)
            shmem[tid] += shmem[tid + s];
        
        __syncthreads();
    }

    if (threadIdx.x == 0)
        atomicAdd(&M[iter], shmem[0]/(n*n*n));
}

/**************************/
/*  Helper C++ Functions  */
/**************************/


/* void callMCIteration2d
 *
 * Location: HOST
 * 
 * This host function makes a call to cudaMcIteration2dKernel
 * which is the kernel that runs a single monte carlo iteration
 * on the provided lattice with the provided system parameters,
 * and updates the lattice accordingly, all on device memory.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - hiprandGenerator_t cg: a cuRAND generator
 *    - float *rands: array for randodm numbers stored on device
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 * 
 * Output:
 *    - none (void)
 * 
 * */
void callMcIteration3d(signed int *lattice,
                       hiprandGenerator_t cg,
                       float *rands,
                       const int n,
                       const float J,
                       const float h){
    
    int blocks = (n*n*n+THREADS - 1)/THREADS;

    CHECK_CURAND(hiprandGenerateUniform(cg,rands,n*n*n));
    cudaMcIteration3dKernel<0><<<blocks,THREADS>>>(lattice,rands,n,J,h);
    cudaMcIteration3dKernel<1><<<blocks,THREADS>>>(lattice,rands,n,J,h);
}

/* void callCalcHamiltonian3d
 *
 * Location: HOST
 * 
 * This host function makes a call to callCalcHamiltonian2d
 * which is the kernel that takes the lattice configuration
 * and calculates the total Hamiltonion (energy) on the device.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *E: energy trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
void callCalcHamiltonian3d(signed int *lattice,
                           float *E,
                           const int n,
                           const float J,
                           const float h,
                           const int iter) {

    int blocks = (n*n*n+THREADS - 1)/THREADS;
    cudaCalcHamiltonian3dKernel<<<blocks,THREADS>>>(lattice,E,n,J,h,iter);
}


/* void callCalcMagnetization2d
 *
 * Location: HOST
 * 
 * This host function makes a call to callCalcMagnetization2d
 * which is the kernel that takes the lattice configuration
 * and calculates the relative magnetization on the device.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *M: magnetization trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
void callCalcMagnetization3d(signed int *lattice,
                             float *M,
                             const int n,
                             const float J,
                             const float h,
                             const int iter) {

    int blocks = (n*n*n+THREADS - 1)/THREADS;
    cudaCalcMagnetization3dKernel<<<blocks,THREADS>>>(lattice,M,n,J,h,iter);
}

/* void print_lattice
 *
 * Location: HOST
 * 
 * Function to print the lattice configuration from host memory,
 * for debugging purposes mostly.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in HOST memeory
 *    - const int n: number of spins in each dimension
 * 
 * Output:
 *    - none (void)
 * 
 * */
// void print_lattice(signed int *lattice, int n)
// {
//     for (int i = 0; i < n; i++)
//     {
//         for (int j = 0; j < n; j++)
//         {
//             if (lattice[(i*n+j)] == 1)
//             {
//                 std::cout << " " << lattice[(i*n+j)] << " ";
//             }
//             else
//             {
//                 std::cout << lattice[(i*n+j)] << " ";
//             }
            
//         }
//         std::cout << " \n";
//     }
// }