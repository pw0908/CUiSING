#include "hip/hip_runtime.h"
#include "ising.h"

/**************************/
/*      CUDA Kernels      */
/**************************/

/* __global__ void init_lattice_2d
 *
 * Location: DEVICE
 * 
 * This CUDA kernel is run on the device in order to fill the lattice
 * with up and down spins. It takes in an array of random uniformly
 * distributed numbers and fills the lattice with integers, either 1
 * or -1. The filling is done in parallel.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - const float *__restrict__ rands: numbers generated from U(0,1), stored in device memory
 *    - const long long n: number of spins in each dimension
 * 
 * Outputs:
 *    - none (void)
 * 
 */
__global__ void init_lattice_2d(signed int *lattice,
                                const float* __restrict__ rands,
                                const long long n) {
    
    const long long tid = (long long)blockDim.x*blockIdx.x + threadIdx.x;
    if (tid >= n*n) return;
    float r = rands[tid];
    lattice[tid] = (r < 0.5f) ? -1 : 1;
}

/* !!!!!UNFINISHED!!!! */
/* __global__ void cudaMCIteration2dKernel
 *
 * Location: DEVICE
 *
 * This CUDA kernel is run on the device in order to run monte carlo 
 * updates in parallel. The kernel implements a checkerboard algorithm
 * to only attempts flips of non-interacting spins that can be flipped
 * in parallel. The attempted flips are accepted/rejected based on the
 * Metropolis criteria which uses the Boltzmann weight. The lattice is
 * updated directly on the device memory.
 * 
 * Periodic boundary conditions are considered. The thread id is
 * converted from linear indexing to 2d indexing, and then the mod
 * operator (%) is employed to wrap the lattice in both dimensions,
 * enforcing periodic boundary conditions.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - const float *__restrict__ rands: numbers generated from U(0,1), stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 * 
 * Output:
 *    - none (void)
 * 
 * */
template<int sub_lattice>
__global__ void cudaMcIteration2dKernel(signed int *lattice,
                                        const float *__restrict__ rands,
                                        const int n,
                                        const float J,
                                        const float h) {

    const long long tid = (long long)(blockDim.x)*blockIdx.x + threadIdx.x;
    const int i = tid / n;
    const int j = tid % n;
    
    if (tid >= n*n)
    {
        return;
    }
    else if ( (i%2 != j%2) != sub_lattice)
    {
        return;
    }

    const int sl = lattice[i*n+(n+(j-1)%n)%n];
    const int sr = lattice[i*n+(n+(j+1)%n)%n];
    const int su = lattice[(n+(i+1)%n)%n*n+j];
    const int sd = lattice[(n+(i-1)%n)%n*n+j];

    const int sum_spins = sl + sr + su + sd;
    const int sij = lattice[tid];
    float boltz = exp(-2.0*sij*(sum_spins*J+h));
    if (rands[tid]<=boltz) lattice[tid] = -sij;
}


/* __global__ void cudaCalcHamiltonian2dKernel
 *
 * Location: DEVICE
 * 
 * This CUDA kernel is run on the device in order to calculate the
 * total Hamiltonian (energy) rapidly. The kernel utilizes shared
 * memory and a binary recution, followed by an atomic addition
 * to maximize the speed of the sum. The local energy of each spin
 * is calculated per thread.
 * 
 * Periodic boundary conditions are considered. The thread id is
 * converted from linear indexing to 2d indexing, and then the mod
 * operator (%) is employed to wrap the lattice in both dimensions,
 * enforcing periodic boundary conditions.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *E: energy trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
__global__ void cudaCalcHamiltonian2dKernel(signed int *lattice,
                                            float *E,
                                            const int n,
                                            const float J,
                                            const float h,
                                            const int iter) {
    
    extern __shared__ float shmem[];

    unsigned tid = threadIdx.x;
    unsigned idx = tid + blockIdx.x * blockDim.x;
    shmem[tid] = 0.0;
    for (; idx < n*n; idx += blockDim.x * gridDim.x)
    {
        unsigned i = idx / n;
        unsigned j = idx % n;
        int sl = lattice[i*n+(n+(j-1)%n)%n];
        int sr = lattice[i*n+(n+(j+1)%n)%n];
        int su = lattice[(n+(i+1)%n)%n*n+j];
        int sd = lattice[(n+(i-1)%n)%n*n+j];
        int sij = lattice[idx];
        
        shmem[tid] -= sij*(J*(sl+sr+su+sd)/2.0+h);
    }
    __syncthreads();

    for (unsigned s = blockDim.x / 2; s>0; s>>=1)
    {
        if (tid < s)
            shmem[tid] += shmem[tid + s];
        
        __syncthreads();
    }

    if (threadIdx.x == 0)
        atomicAdd(&E[iter], shmem[0]/(n*n));
}


/* __global__ void cudaCalcMagnetization2dKernel
 *
 * Location: DEVICE
 * 
 * This CUDA kernel is run on the device in order to calculate the
 * total relative magnetization rapidly. The kernel utilizes shared
 * memory and a binary recution, followed by an atomic addition
 * to maximize the speed of the sum. Relative magnetization is the
 * sum of all spin values divided by the total number of spins.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *M: magnetization trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
__global__ void cudaCalcMagnetization2dKernel(signed int *lattice,
                                              float *M,
                                              const int n,
                                              const float J,
                                              const float h,
                                              const int iter) {
    
    extern __shared__ float shmem[];

    unsigned tid = threadIdx.x;
    unsigned idx = tid + blockIdx.x * blockDim.x;
    shmem[tid] = 0.0;
    for (; idx < n*n; idx += blockDim.x * gridDim.x)
    {
        shmem[tid] += lattice[idx];
    }
    __syncthreads();

    for (unsigned s = blockDim.x / 2; s>0; s>>=1)
    {
        if (tid < s)
            shmem[tid] += shmem[tid + s];
        
        __syncthreads();
    }

    if (threadIdx.x == 0)
        atomicAdd(&M[iter], shmem[0]/(n*n));
}

/**************************/
/*  Helper C++ Functions  */
/**************************/

/* void gen_rands
 *
 * Location: HOST
 * 
 * This host function makes a call to cuRAND to generate pseudorandom
 * numbers drawn from U(0,1) on the device.
 * 
 * Inputs:
 *    - hiprandGenerator_t cg: a cuRAND generator
 *    - float *rands: array for randodm numbers stored on device
 *    - const int n: number of spins in each dimension
 * 
 * Output:
 *    - none (void)
 * 
 * */
void gen_rands(hiprandGenerator_t cg, float *rands, int n)
{
    CHECK_CURAND(hiprandGenerateUniform(cg,rands,n*n));
}


/* void callMCIteration2d
 *
 * Location: HOST
 * 
 * This host function makes a call to cudaMcIteration2dKernel
 * which is the kernel that runs a single monte carlo iteration
 * on the provided lattice with the provided system parameters,
 * and updates the lattice accordingly, all on device memory.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - hiprandGenerator_t cg: a cuRAND generator
 *    - float *rands: array for randodm numbers stored on device
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 * 
 * Output:
 *    - none (void)
 * 
 * */
void callMcIteration2d(signed int *lattice,
                       hiprandGenerator_t cg,
                       float *rands,
                       const int n,
                       const float J,
                       const float h){
    
    int blocks = (n*n+THREADS - 1)/THREADS;

    CHECK_CURAND(hiprandGenerateUniform(cg,rands,n*n));
    cudaMcIteration2dKernel<0><<<blocks,THREADS>>>(lattice,rands,n,J,h);
    cudaMcIteration2dKernel<1><<<blocks,THREADS>>>(lattice,rands,n,J,h);
}

/* void callCalcHamiltonian2d
 *
 * Location: HOST
 * 
 * This host function makes a call to callCalcHamiltonian2d
 * which is the kernel that takes the lattice configuration
 * and calculates the total Hamiltonion (energy) on the device.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *E: energy trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
void callCalcHamiltonian2d(signed int *lattice,
                           float *E,
                           const int n,
                           const float J,
                           const float h,
                           const int iter) {

    int blocks = (n*n+THREADS - 1)/THREADS;
    cudaCalcHamiltonian2dKernel<<<blocks,THREADS>>>(lattice,E,n,J,h,iter);
}


/* void callCalcMagnetization2d
 *
 * Location: HOST
 * 
 * This host function makes a call to callCalcMagnetization2d
 * which is the kernel that takes the lattice configuration
 * and calculates the relative magnetization on the device.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in device memory
 *    - float *M: magnetization trajectory stored in device memory
 *    - const int n: number of spins in each dimension
 *    - const float J: the spin-spin interaction strength
 *    - const float h: the spin-field interaction strength
 *    - const int iter: the current monte carlo iteration
 * 
 * Output:
 *    - none (void)
 * 
 * */
void callCalcMagnetization2d(signed int *lattice,
                             float *M,
                             const int n,
                             const float J,
                             const float h,
                             const int iter) {

    int blocks = (n*n+THREADS - 1)/THREADS;
    cudaCalcMagnetization2dKernel<<<blocks,THREADS>>>(lattice,M,n,J,h,iter);
}

/* void print_lattice
 *
 * Location: HOST
 * 
 * Function to print the lattice configuration from host memory,
 * for debugging purposes mostly.
 * 
 * Inputs:
 *    - signed int *lattice: the lattice stored in HOST memeory
 *    - const int n: number of spins in each dimension
 * 
 * Output:
 *    - none (void)
 * 
 * */
void print_lattice(signed int *lattice, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (lattice[(i*n+j)] == 1)
            {
                std::cout << " " << lattice[(i*n+j)] << " ";
            }
            else
            {
                std::cout << lattice[(i*n+j)] << " ";
            }
            
        }
        std::cout << " \n";
    }
}

/* void writeEM
 *
 * Location: HOST
 * 
 * Function to write the energy (E) and magnetization (M) trajectories
 * to a file in the root directoy, called 'EM.dat'.
 * 
 * Inputs:
 *    - float *E_h: the energy trajectory stored on host memory
 *    - float *M_h: the magnetization trajectory stored on host memory
 *    - const int n_iters: the number of MC iterations
 * 
 * Output:
 *    - none (void)
 * 
 * */
void writeEM(float *E_h, float *M_h, const int n_iters)
{
    std::ofstream emStream;
    std::string emFile = "EM.dat";
    emStream.open(emFile.c_str(), std::fstream::out | std::ofstream::trunc);

    for (unsigned i = 0; i < n_iters; i++)
    {
        if (!(i%10))
        {
            emStream << i << " ";
            emStream << std::fixed << std::setprecision(5) << E_h[i] << " ";
            emStream << std::fixed << std::setprecision(5) << M_h[i] << " \n";
        }
        
    }

}


/* Alas: the main event */
int main(int argc, char **argv){


    /* Default System Parameters */
    unsigned n_iters = 1000;
    unsigned d = 2;
    unsigned n = 100;
    double J = 1.0;
    double h = 0.0;
    
    /* Parse input arguments from the command line */
    if (argc > 5)
    {
        n_iters = atof(argv[1]);
        d = atof(argv[2]);
        n = atof(argv[3]);
        J = atof(argv[4]);
        h = atof(argv[5]);
    }
    else if (argc > 1)
    {
        std::cerr << "Need to supply 5 arguments (or none): int n_iters, int n, double J, double h" << std::endl;
        std::exit(0);
    }

    /* Print a window with selected system parameters */
    std::cout << "=========================" << std::endl;
    std::cout << "n_iters:   " << n_iters    << std::endl;
    std::cout << "d:         " << d          << std::endl;
    std::cout << "n:         " << n          << std::endl;
    std::cout << "J:         " << J          << std::endl;
    std::cout << "h:         " << h          << std::endl;
    std::cout << "=========================" << std::endl;

    /* Create and allocate E and M trajectories on DEVICE */
    float *E;
    float *M;
    CHECK_CUDA(hipMalloc(&E, n_iters * sizeof(*E)));
    CHECK_CUDA(hipMalloc(&M, n_iters * sizeof(*M)));

    /* Create and allocate E_h and M_h trajectories on HOST */
    float *E_h;
    float *M_h;
    E_h = new float[n_iters];
    M_h = new float[n_iters];

    /* rng seeding, based on system time */
    time_t seed;
    time(&seed); // store system time in seed
    srand((unsigned int) seed); // seed random number generator with system time
    
    /* Setup cuRAND and allocated device memory for storing random numbers */
    hiprandGenerator_t cg;
    CHECK_CURAND(hiprandCreateGenerator(&cg, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
    CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(cg, seed));
    float *rands;
    CHECK_CUDA(hipMalloc(&rands, n * n * sizeof(*rands)));

    /* Generate initial random numbers on device, for initializing lattice */
    gen_rands(cg,rands,n);

    /* Setup lattice and allocate memory for storage */
    signed int *lattice;
    CHECK_CUDA(hipMalloc(&lattice, n * n * sizeof(*lattice)));
    /* Initialize the lattice using the previously generated random array on device */
    int blocks = (n*n+THREADS - 1)/THREADS;
    init_lattice_2d<<<blocks,THREADS>>>(lattice,rands, n);

    /* Make sure the device is synced */
    CHECK_CUDA(hipDeviceSynchronize());

    /* initialize timer for monte carlo loop */
    auto t0 = std::chrono::high_resolution_clock::now();

    /* Enter monte carlo loop for n_iters iterations */
    for (int i = 0; i < n_iters; i++)
    {
        callMcIteration2d(lattice,cg,rands,n,J,h);  // update the lattice
        callCalcHamiltonian2d(lattice,E,n,J,h,i);   // calculate the energy and store in E
        callCalcMagnetization2d(lattice,M,n,J,h,i); // calculate the mag and store in M
    }

    /* Ensure device is synced */
    CHECK_CUDA(hipDeviceSynchronize());

    /* End timer for monte carlo and calculate program duration */
    auto t1 = std::chrono::high_resolution_clock::now();
    double duration = (double) std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count();

    /* Copy the E and M trajectory to host for writing */
    CHECK_CUDA(hipMemcpy(E_h, E, n_iters*sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(M_h, M, n_iters*sizeof(float), hipMemcpyDeviceToHost));
    writeEM(E_h,M_h,n_iters); // write traj to file EM.dat

    /* Copy the final latice configuration to the host */
    signed int *lattice_h;
    lattice_h = new signed int [n*n];
    CHECK_CUDA(hipMemcpy(lattice_h, lattice, n*n * sizeof(float), hipMemcpyDeviceToHost));

    /* Output program duration */
    printf("Total Program Time: %f seconds\n", duration * 1e-6);

    
    /* Cleaning up */
    CHECK_CURAND(hiprandDestroyGenerator(cg));
    CHECK_CUDA(hipFree(rands));
    CHECK_CUDA(hipFree(lattice));
    CHECK_CUDA(hipFree(E));
    CHECK_CUDA(hipFree(M));
    delete[] lattice_h;
    delete[] E_h;
    delete[] M_h;

    return 0;
}
